#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

__global__ void bgr_bgra_kernel(char* destImage,
	char* srcImage,
	int* dstPitch,
	int* dstHeigth,
	int srcPitch,
	int srcHeigth
)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride = blockDim.x * gridDim.x;

	int resolution = srcPitch * srcHeigth;
	for (unsigned int i = index; i < resolution; i += stride) {
		destImage[4 * i + 0] = srcImage[i * 3 + 2];
		destImage[4 * i + 1] = srcImage[i * 3 + 1];
		destImage[4 * i + 2] = srcImage[i * 3 + 0];
		destImage[4 * i + 3] = 255;
	}
	return;
}
extern "C" {
	void bgr_bgra(char* rgbaBuffer,
		char* srcBuffer,
		int* dstPitch,
		int* dstHeigth,
		int srcPitch,
		int srcHeigth
	)
	{
		bgr_bgra_kernel << <320, 180 >> > (rgbaBuffer, srcBuffer, dstPitch, dstHeigth, srcPitch, srcHeigth);
		hipDeviceSynchronize();
	}
}
